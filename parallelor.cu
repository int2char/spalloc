#include "hip/hip_runtime.h"

/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include"pathalg.h"
static const int WORK_SIZE = 256;
__global__ void BFShigh(int t,int *m,int index,epair*nei,int *d,int *chan,int edgesize,int tedgesize,int round,int pnodenum)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if(i>=tedgesize)return;
	int from=nei[i].f;
	if (chan[from]<0)return;
	chan[from]=-1;
	int to=nei[i].t;
	d[to]=round;
	if((to%pnodenum)==t)*m=1;
}
__global__ void BFShighN(int t,int *m,int index,epair*nei,int* duan,int*beg,int *d,int *chan,int round,int pnodenum,int nodenum)
{
	int from=threadIdx.x + blockIdx.x*blockDim.x;
	if(from>=nodenum)return;
	if (chan[from]<0)return;
	for(int k=beg[from];k<(beg[from]+duan[from]);k++)
	{
		int to=nei[k].t;
		d[to]=round;
		if((to%pnodenum)==t)*m=1;
	}
}
__global__ void initchan(int s,int *chan,int *d,int *pred,int nodenum)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if(i>=nodenum)return;
	chan[i]=(i==s)?1:-1;
	d[i]=(i==s)?0:inf;
	pred[i]=d[i];
}
__global__ void chanchan(int *m,int *pred,int *d,int *chan,int totalsize,int nodenum)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if(i>=totalsize)return;
	chan[i]=-1;
	if(d[i]<pred[i])
	{
		chan[i]=1;
		pred[i]=d[i];
	}
}
void parallelor::copydata(int s,vector<edge>&edges,int nodenum){
	memset(pre,-1,sizeof(int)*nodenum);
	*m=0;
	for(int i=0;i<nodenum;i++)
		d[i]=INT_MAX/2;
	d[s]=0;
	for(int i=0;i<edges.size();i++)
		aedges[i]=edges[i];
	hipMemcpy(dev_edges,aedges,edges.size()* sizeof(edge),hipMemcpyHostToDevice);
	hipMemcpy(dev_m,m,sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_d,d,sizeof(int)*nodenum,hipMemcpyHostToDevice);
	hipMemcpy(dev_pre,pre,sizeof(int)*nodenum,hipMemcpyHostToDevice);
};
void parallelor::dellocate(){
	delete[]d;
	delete[]pre;
	delete[]aedges;
	delete m;
	hipFree(dev_edges);
	hipFree(dev_m);
	hipFree(dev_d);
	hipFree(dev_pre);
};
void parallelor::allocate(int maxn,int maxedge){
	m=new int;
	d=new int[maxn],pre=new int[maxn];
	aedges=new edge[maxedge];
	hipMalloc(&dev_edges, sizeof(edge)*maxedge);
	hipMalloc((void**)&dev_d,maxn*sizeof(int));
	hipMalloc((void**)&dev_pre,maxn*sizeof(int));
	hipMemcpy(duan,dev_duan,duansize*sizeof(int),hipMemcpyDeviceToHost);
	hipMalloc((void**)&dev_m,sizeof(int));
}
bool parallelor::cutcake(int index){

	cout<<"cut "<<index<<endl;
	if(maxbw-(index+1)*10>=0)
		maxbw-=(index+1)*10;
	else
		{
			cout<<"failure"<<endl;
			return false;
		}
	hleveln[index]++;
	return true;

};
void parallelor::topsort()
{
       	cout<<" in top sort "<<endl;
       	cout<<"node num is "<<nodenum<<endl;
       	queue<int>zero;
       	order=new int[nodenum];
       	ordernode=new int[nodenum];
       	for(int i=0;i<nodenum;i++)
       		if(ancestor[i]==0)
       			zero.push(i);
       	int biao=0;
			while(!zero.empty())
			{
				int node=zero.front();
				zero.pop();
				order[node]=biao++;
				ordernode[biao-1]=node;
				for(int i=0;i<neibour[node].size();i++)
				{
					if((--ancestor[aedges[neibour[node][i]].t])==0)
							zero.push(aedges[neibour[node][i]].t);
				}
			}
		cout<<biao<<" "<<nodenum<<endl;
		cout<<"out top"<<endl;
};
void parallelor::init(vector<edge>&extenedges,vector<vector<int>>&relate,ginfo ginf){
	cout<<"in cuda init"<<endl;
	maxbw=500;
	//allocate in cuda
	edgesize=extenedges.size();nodenum=ginf.enodesize;
	edges=extenedges;
	pesize=ginf.pesize;pnodesize=ginf.pnodesize;
	/*cout<<"es "<<edgesize<<"  pes"<<pesize<<endl;
	dsize=ML*nodenum,presize=ML*nodenum;
	neisize=BS*ML*edgesize;
	duansize=nodenum;
	vector<vector<int>>nd(nodenum,vector<int>());
    neibour=nd;
    vector<int>as(nodenum,0);
    ancestor=as;
	for(int i=0;i<edgesize;i++)
		{
			neibour[extenedges[i].s].push_back(i);
			ancestor[extenedges[i].t]++;
		}
	levelnsize=BS;
	hipMalloc(&dev_edges, sizeof(edge)*edgesize);
	hipMalloc((void**)&dev_d,dsize*sizeof(int));
	hipMalloc((void**)&dev_pred,dsize*sizeof(int));
	hipMalloc((void**)&dev_pre,presize*sizeof(int));
	hipMalloc((void**)&dev_chan,presize*sizeof(int));
	hipMalloc((void**)&dev_m,sizeof(int));
	hipMalloc((void**)&dev_choosel,sizeof(int));
	hipMalloc((void**)&dev_nei,neisize*sizeof(epair));
	hipMalloc((void**)&dev_rela,(WD+1)*edgesize*sizeof(int));
	hipMalloc((void**)&dev_rout,WD*sizeof(int));
	hipMalloc((void**)&dev_routn,sizeof(int));
	hipMalloc((void**)&dev_duan,duansize*sizeof(int));
	hipMalloc((void**)&dev_beg,duansize*sizeof(int));
	hipMalloc((void**)&dev_order,nodenum*sizeof(int));
	hipMalloc((void**)&dev_ordernode,nodenum*sizeof(int));
	hipMalloc((void**)&dev_qian,edgesize*sizeof(int));
	hipMalloc((void**)&dev_qsize,nodenum*sizeof(int));
	hipMalloc((void**)&dev_qbeg,nodenum*sizeof(int));
	//new in host ;
	aedges=new edge[edgesize];
	choosel=new int;
	m=new int;
	pred=new int[dsize];
	d=new int[dsize],pre=new int[presize],chan=new int[presize];
	leveln=new int[levelnsize];
	rela=new int[(WD+1)*edgesize];
	nei=new epair[neisize];
	duan=new int[duansize];
	beg=new int[duansize];
	rout=new int[WD];
	for(int i=0;i<WD;i++)
		rout[i]=-1;
	routn=new int;
	//init in host ;
	*m=0;
	*choosel=0;
	memset(pre,-1,sizeof(int)*presize);
	memset(chan,0,sizeof(int)*presize);
	for(int i=0;i<dsize;i++)
		d[i]=inf,pred[i]=inf;
	for(int i=0;i<edgesize;i++)
		aedges[i]=extenedges[i];
	for(int i=0;i<relate.size();i++)
		for(int j=0;j<WD+1;j++)
			if(j<relate[i].size())
				rela[i*WD+j]=relate[i][j];
			else
				rela[i*WD+j]=-1;
	memset(leveln,0,sizeof(int)*levelnsize);
	int h=0;
	topsort();
	vector<vector<int>>vqian(nodenum,vector<int>());
	int g=0;
	for(int i=0;i<BS*ML;i++)
		{
			int t=0;
			for(int j=0;j<nodenum;j++)
				{
					beg[j]=t;
					duan[j]=neibour[ordernode[j]].size();
					t+=neibour[ordernode[j]].size();
					for(int k=0;k<neibour[ordernode[j]].size();k++)
						{
							nei[h].f=ordernode[j];
							nei[h].t=aedges[neibour[ordernode[j]][k]].t;
							if(i==0)
								vqian[nei[h].t].push_back(nei[h].f);
							h++;
						}
				}
		}
	qian=new int[edgesize];
	qsize=new int[nodenum];
	qbeg=new int[nodenum];
	int y=0;
	int ss=0;
	for(int i=0;i<vqian.size();i++)
		{
			qsize[i]=vqian[i].size();
			qbeg[i]=ss;
			for(int j=0;j<vqian[i].size();j++)
				qian[y++]=vqian[i][j];
			ss+=vqian[i].size();
		}
	hipMemcpy(dev_edges,aedges,edgesize* sizeof(edge),hipMemcpyHostToDevice);
	hipMemcpy(dev_m,m,sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_choosel,choosel,sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_d,d,dsize*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_pred,pred,dsize*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_pre,pre,presize*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_chan,chan,presize*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_rela,rela,(WD+1)*edgesize*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_routn,m,sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_rout,rout,WD*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_nei,nei,neisize*sizeof(epair),hipMemcpyHostToDevice);
	hipMemcpy(dev_duan,duan,duansize*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_beg,beg,duansize*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_order,order,nodenum*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_ordernode,ordernode,nodenum*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_qian,qian,edgesize*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_qsize,qsize,nodenum*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_qbeg,qbeg,nodenum*sizeof(int),hipMemcpyHostToDevice);
	//for(int i=0;i<nodenum;i++)
		//cout<<order[i]<<endl;
	vector<int>tmp(levelnsize,0);
	hleveln=tmp;
	cout<<"out init"<<endl;*/
};

parallelor::parallelor()
{
};
vector<int> parallelor:: routalg(int s,int t,int bw)
{
	int zero=0;
	int index=bw/10-1;
	if(hleveln[index]<=0)cutcake(index);
	int max=0;
	int h=10;
	t=ordernode[t];
	s=ordernode[s];
	cout<<"blasting "<<endl;
	//while(true)
	{
		initchan<< <(nodenum/WORK_SIZE)+1, WORK_SIZE >> >(s,dev_chan,dev_d,dev_pred,nodenum);
		int kk=1,gg=8;
		hipMemcpy(dev_m, &zero, sizeof(int), hipMemcpyHostToDevice);
		do{
			/*hipMemcpy(chan,dev_chan,nodenum*sizeof(int), hipMemcpyDeviceToHost);
			int cc=0;
			for(int i=0;i<nodenum;i++)
				if(chan[i]>=0)
					cc++;
			cout<<cc<<endl;*/
			BFShigh << <(edgesize/WORK_SIZE)+1, WORK_SIZE >> >(t,dev_m,index,dev_nei,dev_d,dev_chan,edgesize,edgesize,kk,pnodesize);
			//BFShighN<< <(nodenum/WORK_SIZE)+1, WORK_SIZE >> >(t,dev_m,index,dev_nei,dev_duan,dev_beg,dev_d,dev_chan,kk,pnodesize,nodenum);
			chanchan<< <(nodenum/WORK_SIZE)+1, WORK_SIZE >> >(dev_m,dev_pred,dev_d,dev_chan,nodenum,nodenum);
			hipMemcpy(m, dev_m, sizeof(int), hipMemcpyDeviceToHost);
			kk++;
		}
		while(*m==0);
		cout<<"kk is: "<<kk<<endl;
		/*hipMemcpy(d, dev_d, sizeof(int)*nodenum, hipMemcpyDeviceToHost);
		int k=0;
		while(t<nodenum)
		{
			k++;
			cout<<d[t]<<endl;
			t+=pnodesize;
		}
		cout<<"over "<<endl;
		/*cudagetrout<< <1,1>> >(dev_qian,dev_qsize,dev_qbeg,dev_d,s,t,dev_rout,dev_routn,dev_choosel,1,nodenum,pnodesize);
		hipMemcpy(routn,dev_routn,sizeof(int),hipMemcpyDeviceToHost);
		hipMemcpy(rout,dev_rout,WD*sizeof(int),hipMemcpyDeviceToHost);
		hipMemcpy(choosel,dev_choosel,sizeof(int),hipMemcpyDeviceToHost);
		cout<<"size is "<<*routn<<endl;
		if(*routn==0)
			{
				if(!cutcake(index))
					return vector<int>();
			}
		else
		{
			cout<<(index+1)*10<<"/"<<*choosel<<": ";
			for(int i=0;i<*routn;i++)
				cout<<rout[i]<<" ";
			cout<<endl;
			return vector<int>();
		}*/
	}
	return vector<int>();
};
int fls(int x)
{
	int position;
	int i;
	if(x!=0)
		for(i=(x>>1),position=0;i!=0;++position)
			i>>=1;
	else
		position=-1;
	return pow(2,position+1);
}
__global__ void push(int*dev_h,int*dev_v,int*dev_ev,int*dev_s,int*dev_t,int E,int W,int *mark)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	int id=threadIdx.x;
	__shared__  int flows[WORK_SIZE];
	__shared__  int biao[WORK_SIZE];
	int eid=i/W;
	if(eid>E)return;
	int offset=i%W;
	biao[id]=offset;
	int s=dev_s[eid]+offset;
	int t=dev_t[eid]+offset;
	if(dev_ev[eid]>0&&dev_v[s]>0)//&&dev_h[s]==dev_h[t]+1)
		flows[id]=1,dev_ev[eid]*=-1;
	if(dev_ev[eid]<0&&dev_v[t]>0&&dev_h[s]+1==dev_h[t])
		flows[id]=2,dev_ev[eid]*=-1;
	int start=(id/W)*W;
	for(int d=W;d>1;d=d/2)
	{
		if(id-start<d/2)
			if(flows[id]<flows[id+d/2])
				flows[id]=flows[id+d/2],biao[id]=biao[id+d/2];
	}
	if(i%W==0)
	{
		if(flows[id]==1)
			dev_v[t+biao[id]]++,*mark=1;
		if(flows[id]==2)
			dev_v[s+biao[id]]++,*mark=1;
	}
	//atomicAdd(mark,1);
	//*mark=1;
};
__global__ void relable(int*dev_h,int*dev_v,int*dev_ev,int*dev_es,int*dev_et)
{

};
void parallelor::prepush(int s,int t,int bw)
{
	cout<<"prepush ing"<<endl;
	int W=fls(WD+1);
	int*dev_mark,*dev_h,*dev_v,*dev_ev,*dev_es,*dev_et;
	int*h=new int[W*pnodesize];
	int*v=new int[W*pnodesize];
	int pesize=edges.size();
	int*ev=new int[pesize];
	int*es=new int[pesize];
	int*et=new int[pesize];
	int*mark=new int;
	*mark=0;
	for(int i=0;i<edges.size();i++)
	{
		ev[i]=1;
		es[i]=edges[i].s;
		et[i]=edges[i].t;
	}
	for(int i=0;i<W*pnodesize;i++)
	{
		h[i]=0;
		v[i]=1;
	}
	for(int i=s*W;i<s*W+WD;i++)
	{
		h[i]=W*pnodesize;
	}
	hipMalloc((void**)&dev_h,W*pnodesize*sizeof(int));
	hipMalloc((void**)&dev_mark,sizeof(int));
	hipMalloc((void**)&dev_v,W*pnodesize*sizeof(int));
	hipMalloc((void**)&dev_ev,pesize*sizeof(int));
	hipMalloc((void**)&dev_es,pesize*sizeof(int));
	hipMalloc((void**)&dev_et,pesize*sizeof(int));
	//hipMemcpy(mark,dev_mark,sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_h,h,W*pnodesize*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_v,v,W*pnodesize*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_ev,ev,pesize*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_et,et,pesize*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_es,es,pesize*sizeof(int),hipMemcpyHostToDevice);
	int block=W*pnodesize/WORK_SIZE;
	int E=edges.size();
	cout<<"node num is: "<<W*pnodesize<<endl;
	for(int i=0;i<10;i++)
	{
		*mark=0;
		hipMemcpy(dev_mark,mark,sizeof(int),hipMemcpyHostToDevice);
		push<< <block+1,WORK_SIZE >> >(dev_h,dev_v,dev_ev,dev_es,dev_et,E,W,dev_mark);
		hipMemcpy(mark,dev_mark,sizeof(int),hipMemcpyDeviceToHost);
		cout<<"mark is :"<<*mark<<endl;
	}
	cout<<"finished"<<endl;

};
