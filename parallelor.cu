#include "hip/hip_runtime.h"

/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include"pathalg.h"
static const int WORK_SIZE = 256;
__global__ void BFShigh(int t,int *m,int index,epair*nei,int *d,int *chan,int edgesize,int tedgesize,int round,int pnodenum)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if(i>=tedgesize)return;
	int from=nei[i].f;
	if (chan[from]<0)return;
	chan[from]=-1;
	int to=nei[i].t;
	d[to]=round;
	if((to%pnodenum)==t)*m=1;
}
__global__ void BFShighN(int t,int *m,int index,epair*nei,int* duan,int*beg,int *d,int *chan,int round,int pnodenum,int nodenum)
{
	int from=threadIdx.x + blockIdx.x*blockDim.x;
	if(from>=nodenum)return;
	if (chan[from]<0)return;
	for(int k=beg[from];k<(beg[from]+duan[from]);k++)
	{
		int to=nei[k].t;
		d[to]=round;
		if((to%pnodenum)==t)*m=1;
	}
}
__global__ void initchan(int s,int *chan,int *d,int *pred,int nodenum)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if(i>=nodenum)return;
	chan[i]=(i==s)?1:-1;
	d[i]=(i==s)?0:inf;
	pred[i]=d[i];
}
__global__ void chanchan(int *m,int *pred,int *d,int *chan,int totalsize,int nodenum)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if(i>=totalsize)return;
	chan[i]=-1;
	if(d[i]<pred[i])
	{
		chan[i]=1;
		pred[i]=d[i];
	}
}
void parallelor::copydata(int s,vector<edge>&edges,int nodenum){
	memset(pre,-1,sizeof(int)*nodenum);
	*m=0;
	for(int i=0;i<nodenum;i++)
		d[i]=INT_MAX/2;
	d[s]=0;
	for(int i=0;i<edges.size();i++)
		aedges[i]=edges[i];
	hipMemcpy(dev_edges,aedges,edges.size()* sizeof(edge),hipMemcpyHostToDevice);
	hipMemcpy(dev_m,m,sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_d,d,sizeof(int)*nodenum,hipMemcpyHostToDevice);
	hipMemcpy(dev_pre,pre,sizeof(int)*nodenum,hipMemcpyHostToDevice);
};
void parallelor::dellocate(){
	/*delete[]d;
	delete[]pre;
	delete[]aedges;
	delete m;
	hipFree(dev_edges);
	hipFree(dev_m);
	hipFree(dev_d);
	hipFree(dev_pre);*/
};
void parallelor::allocate(int maxn,int maxedge){
	m=new int;
	d=new int[maxn],pre=new int[maxn];
	aedges=new edge[maxedge];
	hipMalloc(&dev_edges, sizeof(edge)*maxedge);
	hipMalloc((void**)&dev_d,maxn*sizeof(int));
	hipMalloc((void**)&dev_pre,maxn*sizeof(int));
	hipMemcpy(duan,dev_duan,duansize*sizeof(int),hipMemcpyDeviceToHost);
	hipMalloc((void**)&dev_m,sizeof(int));
}
bool parallelor::cutcake(int index){

	cout<<"cut "<<index<<endl;
	if(maxbw-(index+1)*10>=0)
		maxbw-=(index+1)*10;
	else
		{
			cout<<"failure"<<endl;
			return false;
		}
	hleveln[index]++;
	return true;

};
void parallelor::topsort()
{
       	cout<<" in top sort "<<endl;
       	cout<<"node num is "<<nodenum<<endl;
       	queue<int>zero;
       	order=new int[nodenum];
       	ordernode=new int[nodenum];
       	for(int i=0;i<nodenum;i++)
       		if(ancestor[i]==0)
       			zero.push(i);
       	int biao=0;
			while(!zero.empty())
			{
				int node=zero.front();
				zero.pop();
				order[node]=biao++;
				ordernode[biao-1]=node;
				for(int i=0;i<neibour[node].size();i++)
				{
					if((--ancestor[aedges[neibour[node][i]].t])==0)
							zero.push(aedges[neibour[node][i]].t);
				}
			}
		cout<<biao<<" "<<nodenum<<endl;
		cout<<"out top"<<endl;
};
void parallelor::init(vector<edge>&extenedges,vector<vector<int>>&relate,ginfo ginf){
	cout<<"in cuda init"<<endl;
	maxbw=500;
	//allocate in cuda
	edgesize=extenedges.size();nodenum=ginf.enodesize;
	edges=extenedges;
	pesize=ginf.pesize;pnodesize=ginf.pnodesize;
	/*cout<<"es "<<edgesize<<"  pes"<<pesize<<endl;
	dsize=ML*nodenum,presize=ML*nodenum;
	neisize=BS*ML*edgesize;
	duansize=nodenum;
	vector<vector<int>>nd(nodenum,vector<int>());
    neibour=nd;
    vector<int>as(nodenum,0);
    ancestor=as;
	for(int i=0;i<edgesize;i++)
		{
			neibour[extenedges[i].s].push_back(i);
			ancestor[extenedges[i].t]++;
		}
	levelnsize=BS;
	hipMalloc(&dev_edges, sizeof(edge)*edgesize);
	hipMalloc((void**)&dev_d,dsize*sizeof(int));
	hipMalloc((void**)&dev_pred,dsize*sizeof(int));
	hipMalloc((void**)&dev_pre,presize*sizeof(int));
	hipMalloc((void**)&dev_chan,presize*sizeof(int));
	hipMalloc((void**)&dev_m,sizeof(int));
	hipMalloc((void**)&dev_choosel,sizeof(int));
	hipMalloc((void**)&dev_nei,neisize*sizeof(epair));
	hipMalloc((void**)&dev_rela,(WD+1)*edgesize*sizeof(int));
	hipMalloc((void**)&dev_rout,WD*sizeof(int));
	hipMalloc((void**)&dev_routn,sizeof(int));
	hipMalloc((void**)&dev_duan,duansize*sizeof(int));
	hipMalloc((void**)&dev_beg,duansize*sizeof(int));
	hipMalloc((void**)&dev_order,nodenum*sizeof(int));
	hipMalloc((void**)&dev_ordernode,nodenum*sizeof(int));
	hipMalloc((void**)&dev_qian,edgesize*sizeof(int));
	hipMalloc((void**)&dev_qsize,nodenum*sizeof(int));
	hipMalloc((void**)&dev_qbeg,nodenum*sizeof(int));
	//new in host ;
	aedges=new edge[edgesize];
	choosel=new int;
	m=new int;
	pred=new int[dsize];
	d=new int[dsize],pre=new int[presize],chan=new int[presize];
	leveln=new int[levelnsize];
	rela=new int[(WD+1)*edgesize];
	nei=new epair[neisize];
	duan=new int[duansize];
	beg=new int[duansize];
	rout=new int[WD];
	for(int i=0;i<WD;i++)
		rout[i]=-1;
	routn=new int;
	//init in host ;
	*m=0;
	*choosel=0;
	memset(pre,-1,sizeof(int)*presize);
	memset(chan,0,sizeof(int)*presize);
	for(int i=0;i<dsize;i++)
		d[i]=inf,pred[i]=inf;
	for(int i=0;i<edgesize;i++)
		aedges[i]=extenedges[i];
	for(int i=0;i<relate.size();i++)
		for(int j=0;j<WD+1;j++)
			if(j<relate[i].size())
				rela[i*WD+j]=relate[i][j];
			else
				rela[i*WD+j]=-1;
	memset(leveln,0,sizeof(int)*levelnsize);
	int h=0;
	topsort();
	vector<vector<int>>vqian(nodenum,vector<int>());
	int g=0;
	for(int i=0;i<BS*ML;i++)
		{
			int t=0;
			for(int j=0;j<nodenum;j++)
				{
					beg[j]=t;
					duan[j]=neibour[ordernode[j]].size();
					t+=neibour[ordernode[j]].size();
					for(int k=0;k<neibour[ordernode[j]].size();k++)
						{
							nei[h].f=ordernode[j];
							nei[h].t=aedges[neibour[ordernode[j]][k]].t;
							if(i==0)
								vqian[nei[h].t].push_back(nei[h].f);
							h++;
						}
				}
		}
	qian=new int[edgesize];
	qsize=new int[nodenum];
	qbeg=new int[nodenum];
	int y=0;
	int ss=0;
	for(int i=0;i<vqian.size();i++)
		{
			qsize[i]=vqian[i].size();
			qbeg[i]=ss;
			for(int j=0;j<vqian[i].size();j++)
				qian[y++]=vqian[i][j];
			ss+=vqian[i].size();
		}
	hipMemcpy(dev_edges,aedges,edgesize* sizeof(edge),hipMemcpyHostToDevice);
	hipMemcpy(dev_m,m,sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_choosel,choosel,sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_d,d,dsize*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_pred,pred,dsize*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_pre,pre,presize*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_chan,chan,presize*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_rela,rela,(WD+1)*edgesize*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_routn,m,sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_rout,rout,WD*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_nei,nei,neisize*sizeof(epair),hipMemcpyHostToDevice);
	hipMemcpy(dev_duan,duan,duansize*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_beg,beg,duansize*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_order,order,nodenum*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_ordernode,ordernode,nodenum*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_qian,qian,edgesize*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_qsize,qsize,nodenum*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_qbeg,qbeg,nodenum*sizeof(int),hipMemcpyHostToDevice);
	//for(int i=0;i<nodenum;i++)
		//cout<<order[i]<<endl;
	vector<int>tmp(levelnsize,0);
	hleveln=tmp;
	cout<<"out init"<<endl;*/
};
parallelor::parallelor()
{

};
vector<int> parallelor:: routalg(int s,int t,int bw)
{
	int zero=0;
	int index=bw/10-1;
	if(hleveln[index]<=0)cutcake(index);
	int max=0;
	int h=10;
	t=ordernode[t];
	s=ordernode[s];
	cout<<"blasting "<<endl;
	//while(true)
	{
		initchan<< <(nodenum/WORK_SIZE)+1, WORK_SIZE >> >(s,dev_chan,dev_d,dev_pred,nodenum);
		int kk=1,gg=8;
		hipMemcpy(dev_m, &zero, sizeof(int), hipMemcpyHostToDevice);
		do{
			/*hipMemcpy(chan,dev_chan,nodenum*sizeof(int), hipMemcpyDeviceToHost);
			int cc=0;
			for(int i=0;i<nodenum;i++)
				if(chan[i]>=0)
					cc++;
			cout<<cc<<endl;*/
			BFShigh << <(edgesize/WORK_SIZE)+1, WORK_SIZE >> >(t,dev_m,index,dev_nei,dev_d,dev_chan,edgesize,edgesize,kk,pnodesize);
			//BFShighN<< <(nodenum/WORK_SIZE)+1, WORK_SIZE >> >(t,dev_m,index,dev_nei,dev_duan,dev_beg,dev_d,dev_chan,kk,pnodesize,nodenum);
			chanchan<< <(nodenum/WORK_SIZE)+1, WORK_SIZE >> >(dev_m,dev_pred,dev_d,dev_chan,nodenum,nodenum);
			hipMemcpy(m, dev_m, sizeof(int), hipMemcpyDeviceToHost);
			kk++;
		}
		while(*m==0);
		cout<<"kk is: "<<kk<<endl;
		/*hipMemcpy(d, dev_d, sizeof(int)*nodenum, hipMemcpyDeviceToHost);
		int k=0;
		while(t<nodenum)
		{
			k++;81
135
1449
2673
4896
			cout<<d[t]<<endl;
			t+=pnodesize;
		}
		cout<<"over "<<endl;
		/*cudagetrout<< <1,1>> >(dev_qian,dev_qsize,dev_qbeg,dev_d,s,t,dev_rout,dev_routn,dev_choosel,1,nodenum,pnodesize);
		hipMemcpy(routn,dev_routn,sizeof(int),hipMemcpyDeviceToHost);
		hipMemcpy(rout,dev_rout,WD*sizeof(int),hipMemcpyDeviceToHost);
		hipMemcpy(choosel,dev_choosel,sizeof(int),hipMemcpyDeviceToHost);
		cout<<"size is "<<*routn<<endl;
		if(*routn==0)
			{
				if(!cutcake(index))
					return vector<int>();
			}
		else
		{
			cout<<(index+1)*10<<"/"<<*choosel<<": ";
			for(int i=0;i<*routn;i++)
				cout<<rout[i]<<" ";
			cout<<endl;
			return vector<int>();
		}*/
	}
	return vector<int>();
};
int fls(int x)
{
	int position;
	int i;
	if(x!=0)
		for(i=(x>>1),position=0;i!=0;++position)
			i>>=1;
	else
		position=-1;
	return pow(2,position+1);
}
/*__global__ void push(int*dev_h,int*dev_v,int*dev_ev,int*dev_s,int*dev_t,int E,int W,int *mark,int sorce,int end)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	int id=threadIdx.x;
	__shared__  int flows[WORK_SIZE];
	__shared__  int biao[WORK_SIZE];
	int eid=i/W;
	if(eid>E)return;
	int offset=i%W;
	biao[id]=offset;
	int s=dev_s[eid]*(W+1)+offset;
	int t=dev_t[eid]*(W+1)+offset;
	if(dev_ev[eid]>0&&dev_v[s]>0&&s!=sorce&&dev_s[eid]!=end)
	{
		if(dev_h[s]==dev_h[t]+1)
			flows[id]=1,dev_ev[eid]*=-1;
	}
	if(dev_ev[eid]<0&&dev_v[t]>0&&t!=sorce&&dev_t[eid]!=end)
	{
		if(dev_h[s]+1==dev_h[t])
			flows[id]=2,dev_ev[eid]*=-1;
	}
	int start=(id/W)*W;
	for(int d=W;d>1;d=d/2)
	{
		if(id-start<d/2)
			if(flows[id]<flows[id+d/2])
				flows[id]=flows[id+d/2],biao[id]=biao[id+d/2];
	}
	if(i%W==0)
	{
		if(flows[id]==1)atomicAdd(&dev_v[t+biao[id]],1),atomicSub(&dev_v[s+biao[id]],1),*mark=1;
		if(flows[id]==2)atomicAdd(&dev_v[s+biao[id]],1),atomicSub(&dev_v[t+biao[id]],1),*mark=1;
	}
};*/

__global__ void push(int*dev_h,int*dev_v,int* dev_esign,int* dev_emark,int*dev_neie,int*dev_nein,int N,int max,int W,int s,int t)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if(i>=N||dev_v[i]==0||i/W==s||i/W==t)return;
	int h=dev_h[i];
	int b=i*max;
	int offset=i%W;
	int value=dev_v[i];
	for(int j=0;j<max;j++)
	{
		if(dev_nein[b+j]<INT_MAX&&value>0)
		{
			if(h==dev_h[dev_nein[b+j]]+1&&dev_neie[b+j]*dev_esign[abs(dev_neie[b+j])]>0)
				{
					if(dev_neie[b+j]>0)
						dev_emark[abs(dev_neie[b+j])]=dev_nein[b+j];
					else
						dev_emark[abs(dev_neie[b+j])]=-i;
					value--;
				}
		}
		else
			break;
	}
};
__global__ void aggregate(int* dev_esign,int*dev_v,int* dev_emark,int W,int E,int*mark)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if(i>=E)return;
	int id=threadIdx.x;
	int eiid=id/W;
	int start=eiid*W;
	int eid=i/W;
	__shared__ int emids[500];
	emids[eiid]=INT_MAX;
	__syncthreads();
	if(dev_emark[i]<INT_MAX)
		emids[eiid]=dev_emark[i];
	__syncthreads();
	int emid=emids[eiid];
	if(id==start&&emid<INT_MAX)
		{
			int s=abs(dev_esign[eid])*(W+1)+abs(emid)%(W+1);
			if(emid>0)
			{	atomicSub(&dev_v[s],1);
				atomicAdd(&dev_v[abs(emid)],1);
				*mark=1;
				dev_esign[eid]*=-1;
			}
			else
			{
				atomicAdd(&dev_v[s],1);
				atomicSub(&dev_v[abs(emid)],1);
				*mark=1;
				dev_esign[eid]*=-1;
			}
		}
	dev_emark[i]=INT_MAX;
};
__global__ void aggregate1(int* dev_esign,int*dev_v,int* dev_emark,int W,int E,int*mark)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if(i>=E)return;
	__shared__  int emark[WORK_SIZE];
	int id=threadIdx.x;
	emark[id]=dev_emark[i];
	int start=(id/W)*W;
	int eid=i/W;
	for(int d=W;d>1;d=d/2)
		if(id-start<d/2)
			if(abs(emark[id])>abs(emark[id+d/2]))
				emark[id]=emark[id+d/2];
	if(id==start&&emark[id]<INT_MAX)
		{
			int s=abs(dev_esign[eid])*(W+1)+abs(emark[id])%(W+1);
			if(emark[id]>0)
			{	atomicSub(&dev_v[s],1);
				atomicAdd(&dev_v[abs(emark[id])],1);
				*mark=1;
				dev_esign[eid]*=-1;
			}
			else
			{
				atomicAdd(&dev_v[s],1);
				atomicSub(&dev_v[abs(emark[id])],1);
				*mark=1;
				dev_esign[eid]*=-1;
			}
		}
	dev_emark[i]=INT_MAX;
};
__global__ void aggregate2(int* dev_esign,int*dev_v,int* dev_emark,int W,int E,int*mark)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if(i>=E)return;
	int emid=dev_emark[i];
	if(emid<INT_MAX)
	{
		int s=abs(dev_esign[i])*(W+1)+abs(emid)%(W+1);
		if(emid>=0)
		{	atomicSub(&dev_v[s],1);
			atomicAdd(&dev_v[abs(emid)],1);
			*mark=1;
			dev_esign[i]*=-1;
		}
		else
		{
			atomicAdd(&dev_v[s],1);
			atomicSub(&dev_v[abs(emid)],1);
			*mark=1;
			dev_esign[i]*=-1;
		}
	}
	dev_emark[i]=INT_MAX;
};
__global__ void relable(int*dev_h,int*dev_v,int N,int*mark,int*dev_nein,int*dev_neie,int *dev_esign,int max,int W,int s,int t)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if(i>=N||dev_v[i]==0||i/W==s||i/W==t)return;
	int b=i*max;
	int mini=INT_MAX;
	for(int j=0;j<max;j++)
	{
		if(dev_nein[b+j]<INT_MAX)
		{
			if(dev_neie[b+j]*dev_esign[abs(dev_neie[b+j])]>0)
				mini=min(mini,dev_h[abs(dev_nein[b+j])]);
		}
		else
			break;
	}
	if(mini!=INT_MAX)
		dev_h[i]=mini+1,*mark=1;

};
void parallelor::prepush(int s,int t,int bw)
{
	cout<<"begin"<<endl;
	int W=WD+1;
	int*dev_h,*h,*dev_v,*v,*dev_neie,*neie,*dev_nein,*nein;
	int*dev_esign,*esign;
	int *dev_emark,*emark,*mark,*dev_mark;
	int *minarray;
	h=new int[W*pnodesize];
	v=new int[W*pnodesize];
	minarray=new int[pnodesize];
	mark=new int;
	vector<vector<int>>rawneie(W*pnodesize,vector<int>());
	vector<vector<int>>rawnein(W*pnodesize,vector<int>());
	for(int i=0;i<edges.size();i++)
		for(int j=0;j<W-1;j++)
			{
				int s=edges[i].s*W+j;
				int t=edges[i].t*W+j;
				rawneie[s].push_back(i);
				rawneie[t].push_back(-i);
				rawnein[s].push_back(t);
				rawnein[t].push_back(s);
			}
	int max=0;
	for(int i=0;i<rawnein.size();i++)
		if(rawnein[i].size()>max)max=rawnein[i].size();
	max++;
	neie=new int[W*pnodesize*max];
	nein=new int[W*pnodesize*max];
	for(int i=0;i<W*pnodesize;i++)
	{
		for(int j=0;j<max;j++)
		{
			if(j<rawneie[i].size())
				{
					neie[i*max+j]=rawneie[i][j];
					nein[i*max+j]=rawnein[i][j];
				}
			else
				{
					neie[i*max+j]=INT_MAX;
					nein[i*max+j]=INT_MAX;
				}
		}
	}
	emark=new int[edges.size()];
	esign=new int[edges.size()];
	for(int i=0;i<edges.size();i++)
		emark[i]=INT_MAX;
	for(int i=0;i<edges.size();i++)
		esign[i]=edges[i].s;
	for(int i=0;i<W*pnodesize;i++)
		{
			h[i]=0;
			v[i]=0;
		}
	for(int i=0;i<edges.size();i++)
		if(edges[i].s==s)
			{
				v[W*edges[i].t+1]=1;
				esign[i]*=-1;
			}
	for(int i=s*W;i<s*W+W;i++)
		h[i]=WD+1;
	hipMalloc((void**)&dev_h,W*pnodesize*sizeof(int));
	hipMalloc((void**)&dev_mark,sizeof(int));
	hipMalloc((void**)&dev_v,W*pnodesize*sizeof(int));
	hipMalloc((void**)&dev_neie,W*max*pnodesize*sizeof(int));
	hipMalloc((void**)&dev_nein,W*max*pnodesize*sizeof(int));
	hipMalloc((void**)&dev_esign,edges.size()*sizeof(int));
	hipMalloc((void**)&dev_emark,edges.size()*sizeof(int));
	hipMemcpy(dev_h,h,W*pnodesize*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_v,v,W*pnodesize*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_mark,mark,sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_neie,neie,W*max*pnodesize*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_nein,nein,W*max*pnodesize*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_esign,esign,edges.size()*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_emark,emark,edges.size()*sizeof(int),hipMemcpyHostToDevice);
	*mark=1;
	int time=0;
	while(*mark>0)
	{
		*mark=0;
		hipMemcpy(dev_mark,mark,sizeof(int),hipMemcpyHostToDevice);
		push<<<W*pnodesize/WORK_SIZE+1,WORK_SIZE>>>(dev_h,dev_v,dev_esign,dev_emark,dev_neie,dev_nein,W*pnodesize,max,W,s,t);
		//aggregate<<<edges.size()*(W-1)/WORK_SIZE+1,WORK_SIZE>>>(dev_esign,dev_v,dev_emark,W-1,edges.size()*(W-1),dev_mark);
		aggregate2<<<edges.size()/WORK_SIZE+1,WORK_SIZE>>>(dev_esign,dev_v,dev_emark,W-1,edges.size(),dev_mark);
		relable<<<W*pnodesize/WORK_SIZE+1,WORK_SIZE>>>(dev_h,dev_v,W*pnodesize,dev_mark,dev_nein,dev_neie,dev_esign,max,W,s,t);
		hipMemcpy(mark,dev_mark,sizeof(int),hipMemcpyDeviceToHost);
		time++;
	}
	cout<<"times is :"<<time<<endl;
	hipMemcpy(v,dev_v,W*pnodesize*sizeof(int),hipMemcpyDeviceToHost);
	hipMemcpy(h,dev_h,W*pnodesize*sizeof(int),hipMemcpyDeviceToHost);
	for(int i=0;i<W*pnodesize;i++)
		if(v[i]!=0)
			cout<<i<<" "<<i/W<<" "<<h[i]<<" "<<v[i]<<endl;
	delete[] h;
	delete[] minarray;
	delete[] v;
	delete[] mark;
	delete[] neie;
	delete[] nein;
	delete[]emark;
	delete[]esign;
	hipFree(dev_h);
	hipFree(dev_mark);
	hipFree(dev_v);
	hipFree(dev_neie);
	hipFree(dev_nein);
	hipFree(dev_esign);
	hipFree(dev_emark);
};
