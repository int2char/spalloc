#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include"pathalg.h"
static const int WORK_SIZE = 512;
__global__ void BFShigh(int t,int *m,int index,epair*nei,int *d,int *chan,int edgesize,int tedgesize,int round,int pnodenum)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if(i>=tedgesize)return;
	int from=nei[i].f;
	if (chan[from]<0)return;
	chan[from]=-1;
	int to=nei[i].t;
	d[to]=round;
	if((to%pnodenum)==t)*m=1;
}
__global__ void BFShighN(int t,int *m,int index,epair*nei,int* duan,int*beg,int *d,int *chan,int round,int pnodenum,int nodenum)
{
	int from=threadIdx.x + blockIdx.x*blockDim.x;
	if(from>=nodenum)return;
	if (chan[from]<0)return;
	for(int k=beg[from];k<(beg[from]+duan[from]);k++)
	{
		int to=nei[k].t;
		d[to]=round;
		if((to%pnodenum)==t)*m=1;
	}
}
__global__ void initchan(int s,int *chan,int *d,int *pred,int nodenum)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if(i>=nodenum)return;
	chan[i]=(i==s)?1:-1;
	d[i]=(i==s)?0:inf;
	pred[i]=d[i];
}
__global__ void chanchan(int *m,int *pred,int *d,int *chan,int totalsize,int nodenum)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if(i>=totalsize)return;
	chan[i]=-1;
	if(d[i]<pred[i])
	{
		chan[i]=1;
		pred[i]=d[i];
	}
}
void parallelor::copydata(int s,vector<edge>&edges,int nodenum){
	memset(pre,-1,sizeof(int)*nodenum);
	*m=0;
	for(int i=0;i<nodenum;i++)
		d[i]=INT_MAX/2;
	d[s]=0;
	for(int i=0;i<edges.size();i++)
		aedges[i]=edges[i];
	hipMemcpy(dev_edges,aedges,edges.size()* sizeof(edge),hipMemcpyHostToDevice);
	hipMemcpy(dev_m,m,sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_d,d,sizeof(int)*nodenum,hipMemcpyHostToDevice);
	hipMemcpy(dev_pre,pre,sizeof(int)*nodenum,hipMemcpyHostToDevice);
};
void parallelor::dellocate(){
	/*delete[]d;
	delete[]pre;
	delete[]aedges;
	delete m;
	hipFree(dev_edges);
	hipFree(dev_m);
	hipFree(dev_d);
	hipFree(dev_pre);*/
};
void parallelor::allocate(int maxn,int maxedge){
	m=new int;
	d=new int[maxn],pre=new int[maxn];
	aedges=new edge[maxedge];
	hipMalloc(&dev_edges, sizeof(edge)*maxedge);
	hipMalloc((void**)&dev_d,maxn*sizeof(int));
	hipMalloc((void**)&dev_pre,maxn*sizeof(int));
	hipMemcpy(duan,dev_duan,duansize*sizeof(int),hipMemcpyDeviceToHost);
	hipMalloc((void**)&dev_m,sizeof(int));
}
bool parallelor::cutcake(int index){

	cout<<"cut "<<index<<endl;
	if(maxbw-(index+1)*10>=0)
		maxbw-=(index+1)*10;
	else
		{
			cout<<"failure"<<endl;
			return false;
		}
	hleveln[index]++;
	return true;

};
void parallelor::topsort()
{
       	cout<<" in top sort "<<endl;
       	cout<<"node num is "<<nodenum<<endl;
       	queue<int>zero;
       	order=new int[nodenum];
       	ordernode=new int[nodenum];
       	for(int i=0;i<nodenum;i++)
       		if(ancestor[i]==0)
       			zero.push(i);
       	int biao=0;
			while(!zero.empty())
			{
				int node=zero.front();
				zero.pop();
				order[node]=biao++;
				ordernode[biao-1]=node;
				for(int i=0;i<neibour[node].size();i++)
				{
					if((--ancestor[aedges[neibour[node][i]].t])==0)
							zero.push(aedges[neibour[node][i]].t);
				}
			}
		cout<<biao<<" "<<nodenum<<endl;
		cout<<"out top"<<endl;
};
void parallelor::init(vector<edge>&extenedges,vector<vector<int>>&relate,ginfo ginf){
	cout<<"in cuda init"<<endl;
	maxbw=500;
	//allocate in cuda
	edgesize=extenedges.size();nodenum=ginf.enodesize;
	edges=extenedges;
	pesize=ginf.pesize;pnodesize=ginf.pnodesize;
	cout<<"out cuda init"<<endl;
	/*cout<<"es "<<edgesize<<"  pes"<<pesize<<endl;
	dsize=ML*nodenum,presize=ML*nodenum;
	neisize=BS*ML*edgesize;
	duansize=nodenum;
	vector<vector<int>>nd(nodenum,vector<int>());
    neibour=nd;
    vector<int>as(nodenum,0);
    ancestor=as;
	for(int i=0;i<edgesize;i++)
		{
			neibour[extenedges[i].s].push_back(i);
			ancestor[extenedges[i].t]++;
		}
	levelnsize=BS;
	hipMalloc(&dev_edges, sizeof(edge)*edgesize);
	hipMalloc((void**)&dev_d,dsize*sizeof(int));
	hipMalloc((void**)&dev_pred,dsize*sizeof(int));
	hipMalloc((void**)&dev_pre,presize*sizeof(int));
	hipMalloc((void**)&dev_chan,presize*sizeof(int));
	hipMalloc((void**)&dev_m,sizeof(int));
	hipMalloc((void**)&dev_choosel,sizeof(int));
	hipMalloc((void**)&dev_nei,neisize*sizeof(epair));
	hipMalloc((void**)&dev_rela,(WD+1)*edgesize*sizeof(int));
	hipMalloc((void**)&dev_rout,WD*sizeof(int));
	hipMalloc((void**)&dev_routn,sizeof(int));
	hipMalloc((void**)&dev_duan,duansize*sizeof(int));
	hipMalloc((void**)&dev_beg,duansize*sizeof(int));
	hipMalloc((void**)&dev_order,nodenum*sizeof(int));
	hipMalloc((void**)&dev_ordernode,nodenum*sizeof(int));
	hipMalloc((void**)&dev_qian,edgesize*sizeof(int));
	hipMalloc((void**)&dev_qsize,nodenum*sizeof(int));
	hipMalloc((void**)&dev_qbeg,nodenum*sizeof(int));
	//new in host ;
	aedges=new edge[edgesize];
	choosel=new int;
	m=new int;
	pred=new int[dsize];
	d=new int[dsize],pre=new int[presize],chan=new int[presize];
	leveln=new int[levelnsize];
	rela=new int[(WD+1)*edgesize];
	nei=new epair[neisize];
	duan=new int[duansize];
	beg=new int[duansize];
	rout=new int[WD];
	for(int i=0;i<WD;i++)
		rout[i]=-1;
	routn=new int;
	//init in host ;
	*m=0;
	*choosel=0;
	memset(pre,-1,sizeof(int)*presize);
	memset(chan,0,sizeof(int)*presize);
	for(int i=0;i<dsize;i++)
		d[i]=inf,pred[i]=inf;
	for(int i=0;i<edgesize;i++)
		aedges[i]=extenedges[i];
	for(int i=0;i<relate.size();i++)
		for(int j=0;j<WD+1;j++)
			if(j<relate[i].size())
				rela[i*WD+j]=relate[i][j];
			else
				rela[i*WD+j]=-1;
	memset(leveln,0,sizeof(int)*levelnsize);
	int h=0;
	topsort();
	vector<vector<int>>vqian(nodenum,vector<int>());
	int g=0;
	for(int i=0;i<BS*ML;i++)
		{
			int t=0;
			for(int j=0;j<nodenum;j++)
				{
					beg[j]=t;
					duan[j]=neibour[ordernode[j]].size();
					t+=neibour[ordernode[j]].size();
					for(int k=0;k<neibour[ordernode[j]].size();k++)
						{
							nei[h].f=ordernode[j];
							nei[h].t=aedges[neibour[ordernode[j]][k]].t;
							if(i==0)
								vqian[nei[h].t].push_back(nei[h].f);
							h++;
						}
				}
		}
	qian=new int[edgesize];
	qsize=new int[nodenum];
	qbeg=new int[nodenum];
	int y=0;
	int ss=0;
	for(int i=0;i<vqian.size();i++)
		{
			qsize[i]=vqian[i].size();
			qbeg[i]=ss;
			for(int j=0;j<vqian[i].size();j++)
				qian[y++]=vqian[i][j];
			ss+=vqian[i].size();
		}
	hipMemcpy(dev_edges,aedges,edgesize* sizeof(edge),hipMemcpyHostToDevice);
	hipMemcpy(dev_m,m,sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_choosel,choosel,sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_d,d,dsize*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_pred,pred,dsize*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_pre,pre,presize*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_chan,chan,presize*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_rela,rela,(WD+1)*edgesize*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_routn,m,sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_rout,rout,WD*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_nei,nei,neisize*sizeof(epair),hipMemcpyHostToDevice);
	hipMemcpy(dev_duan,duan,duansize*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_beg,beg,duansize*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_order,order,nodenum*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_ordernode,ordernode,nodenum*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_qian,qian,edgesize*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_qsize,qsize,nodenum*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_qbeg,qbeg,nodenum*sizeof(int),hipMemcpyHostToDevice);
	//for(int i=0;i<nodenum;i++)
		//cout<<order[i]<<endl;
	vector<int>tmp(levelnsize,0);
	hleveln=tmp;
	cout<<"out init"<<endl;*/
};
parallelor::parallelor()
{

};
vector<int> parallelor:: routalg(int s,int t,int bw)
{
	int zero=0;
	int index=bw/10-1;
	if(hleveln[index]<=0)cutcake(index);
	int max=0;
	int h=10;
	t=ordernode[t];
	s=ordernode[s];
	cout<<"blasting "<<endl;
	while(true)
	for(int i=0;i<1;i++)
	{
		initchan<< <(nodenum/WORK_SIZE)+1, WORK_SIZE >> >(s,dev_chan,dev_d,dev_pred,nodenum);
		int kk=1,gg=8;
		hipMemcpy(dev_m, &zero, sizeof(int), hipMemcpyHostToDevice);
		do{
			/*hipMemcpy(chan,dev_chan,nodenum*sizeof(int), hipMemcpyDeviceToHost);
			int cc=0;
			for(int i=0;i<nodenum;i++)
				if(chan[i]>=0)
					cc++;
			cout<<cc<<endl;*/
			BFShigh << <(edgesize/WORK_SIZE)+1, WORK_SIZE >> >(t,dev_m,index,dev_nei,dev_d,dev_chan,edgesize,edgesize,kk,pnodesize);
			//BFShighN<< <(nodenum/WORK_SIZE)+1, WORK_SIZE >> >(t,dev_m,index,dev_nei,dev_duan,dev_beg,dev_d,dev_chan,kk,pnodesize,nodenum);
			chanchan<< <(nodenum/WORK_SIZE)+1, WORK_SIZE >> >(dev_m,dev_pred,dev_d,dev_chan,nodenum,nodenum);
			hipMemcpy(m, dev_m, sizeof(int), hipMemcpyDeviceToHost);
			kk++;
		}
		while(*m==0);
		cout<<"kk is: "<<kk<<endl;
		/*hipMemcpy(d, dev_d, sizeof(int)*nodenum, hipMemcpyDeviceToHost);
		int k=0;
		while(t<nodenum)
		{
			k++;
			cout<<d[t]<<endl;
			t+=pnodesize;
		}
		cout<<"over "<<endl;
		/*cudagetrout<< <1,1>> >(dev_qian,dev_qsize,dev_qbeg,dev_d,s,t,dev_rout,dev_routn,dev_choosel,1,nodenum,pnodesize);
		hipMemcpy(routn,dev_routn,sizeof(int),hipMemcpyDeviceToHost);
		hipMemcpy(rout,dev_rout,WD*sizeof(int),hipMemcpyDeviceToHost);
		hipMemcpy(choosel,dev_choosel,sizeof(int),hipMemcpyDeviceToHost);
		cout<<"size is "<<*routn<<endl;
		if(*routn==0)
			{
				if(!cutcake(index))
					return vector<int>();
			}
		else
		{
			cout<<(index+1)*10<<"/"<<*choosel<<": ";
			for(int i=0;i<*routn;i++)
				cout<<rout[i]<<" ";
			cout<<endl;
			return vector<int>();
		}*/
	}
	return vector<int>();
};
int fls(int x)
{
	int position;
	int i;
	if(x!=0)
		for(i=(x>>1),position=0;i!=0;++position)
			i>>=1;
	else
		position=-1;
	return pow(2,position+1);
}
__global__ void push(int*dev_h,int*dev_v,int* dev_esign,int* dev_emark,int*dev_neie,int*dev_nein,int N,int max,int W,int s,int t,int*mark)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	int bi=i%N;
	int value=dev_v[i];
	if(i>=N*LY||value==0||bi/W==s||bi/W==t)return;
	int h=dev_h[i];
	int b=i*max;
	int minheight=INT_MAX;
	for(int j=0;j<max;j++)
	{
		int nbj=dev_nein[b+j];
		if(value>0&&nbj<INT_MAX)
		{
			int ebj=dev_neie[b+j];
			int hnbj=dev_h[nbj];
			int eid=abs(ebj)-1;
			if((ebj^dev_esign[eid])>0)
			{
				if(dev_emark[eid]>INT_MAX/2&&h==hnbj+1)
				{
					dev_emark[eid]=(ebj>0)?nbj:i;
					value--;
					*mark=1;
				}
				minheight=min(minheight,hnbj);
			}
		}
		else
			break;
	}
	if(value>0&&minheight<INT_MAX){dev_h[i]=minheight+1;*mark=1;}
};
__global__ void push1(int*dev_h,int*dev_v,int* dev_esign,int* dev_emark,int*dev_neie,int*dev_nein,int N,int max,int W,int s,int t,int*mark)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	int bi=i%N;
	int value=dev_v[i];
	if(i>=N*LY||value==0||bi/W==s||bi/W==t)return;
	int h=dev_h[i];
	int b=i*max;
	int minheight=INT_MAX;
	for(int j=0;j<max;j++)
	{
		int nbj=dev_nein[b+j];
		if(value>0&&nbj<INT_MAX)
		{
			int ebj=dev_neie[b+j];
			int hnbj=dev_h[nbj];
			int eid=abs(ebj)-1;
			if((ebj^dev_esign[eid])>0)
			{
				if(dev_emark[eid]==0&&h==hnbj+1)
				{
					dev_emark[eid]++;
					value--;
					*mark=1;
				}
				minheight=min(minheight,hnbj);
			}
		}
		else
			break;
	}
	if(value>0&&minheight<INT_MAX){dev_h[i]=minheight+1;*mark=1;}
};
__global__ void pushrelable(int*dev_h,int*dev_v,int* dev_esign,int* dev_emark,int*dev_neie,int*dev_nein,int N,int max,int W,int s,int t,int*mark)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	int bi=i%N;
	int value=dev_v[i];
	if(i>=N*LY||value==0||bi/W==s||bi/W==t)return;
	int h=dev_h[i];
	int b=i*max;
	int minheight=INT_MAX;
	for(int j=0;j<max;j++)
	{
		int nbj=dev_nein[b+j];
		if(value>0&&nbj<INT_MAX)
		{
			int ebj=dev_neie[b+j];
			int hnbj=dev_h[nbj];
			int eid=abs(ebj)-1;
			if((ebj^dev_esign[eid])>0)
			{
				if(dev_emark[eid]==i)
				{
					dev_emark[eid]++;
					atomicSub(&dev_v[i],1);
					atomicAdd(&dev_v[nbj],1);
					value--;
					dev_esign[eid]*=-1;
					*mark=1;
				}
				minheight=min(minheight,hnbj);
			}
		}
		else
			break;
	}
	if(value>0&&minheight<INT_MAX){dev_h[i]=minheight+1;*mark=1;}
};
__global__ void aggregate3(int* dev_esign,int* dev_v,int* dev_emark,int* dev_st,int* dev_te,int*dev_h,int W,int E)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if(i>=E*LY)return;
		int s,t;
	if(dev_emark[i]>0)
	{
		if(dev_esign[i]>0)
		{
			s=dev_st[i];
			t=dev_te[i]+1;
		}
		else
		{
			t=dev_st[i];
			s=dev_te[i]+1;
		}
		for(int k=0;k<W;k++)
			{
				int h1=dev_h[s+k];
				int h2=dev_h[t+k];
				if(dev_v[s+k]>0&&h1==h2+1)
				{
					atomicSub(&dev_v[s+k],1);
					atomicAdd(&dev_v[t+k],1);
					dev_esign[i]*=-1;
					break;
				}
			}
	}
	dev_emark[i]=0;
};
__global__ void aggregate4(int* dev_esign,int* dev_v,int* dev_emark,int* dev_st,int* dev_te,int*dev_h,int W,int E)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if(i>=E*LY)return;
	int s,t;
	dev_emark[i]=INT_MAX;
	if(dev_esign[i]>0)
	{
		s=dev_st[i];
		t=dev_te[i]+1;
	}
	else
	{
		t=dev_st[i];
		s=dev_te[i]+1;
	}
	for(int k=0;k<W;k++)
		{
			int h1=dev_h[s+k];
			int h2=dev_h[t+k];
			if(dev_v[s+k]>0&&h1==h2+1)
			{
				dev_emark[i]=s+k;
				break;
			}
		}
};
__global__ void aggregate2(int* dev_esign,int*dev_v,int* dev_emark,int W,int E)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if(i>=E*LY)return;
	int emid=dev_emark[i];
	if(emid<INT_MAX)
	{
		int s=abs(dev_esign[i])-2+emid%W;
		if(dev_esign[i]>0)
		{	atomicSub(&dev_v[s],1);
			atomicAdd(&dev_v[emid],1);
		}
		else
		{	atomicAdd(&dev_v[s],1);
			atomicSub(&dev_v[emid],1);
		}
		dev_esign[i]*=-1;
	}
	dev_emark[i]=INT_MAX;
};
__global__ void aggregate5(int* dev_esign,int* dev_v,int* dev_emark,int* dev_st,int* dev_te,int*dev_h,int W,int E)
{
        int i = threadIdx.x + blockIdx.x*blockDim.x;
        if(i>=E*LY*W)return;
        int s,t;
        int eid=i/W;
        int k=i%W;
        if(dev_esign[eid]>0)
        {
                s=dev_st[eid];
                t=dev_te[eid]+1;
        }
        else
        {
                t=dev_st[eid];
                s=dev_te[eid]+1;
        }
        int h1=dev_h[s+k];
        int h2=dev_h[t+k];
        if(dev_v[s+k]>0&&h1==h2+1)
                dev_emark[eid]=s+k;
};

__global__ void relable(int*dev_h,int*dev_v,int N,int*mark,int*dev_nein,int*dev_neie,int *dev_esign,int max,int W,int s,int t)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	int bi=i%N;
	if(i>=N*LY||dev_v[i]==0||bi/W==s||bi/W==t)return;
	int b=i*max;
	int mini=INT_MAX;
	for(int j=0;j<max;j++)
	{
		if(dev_nein[b+j]<INT_MAX)
		{
			if((dev_neie[b+j]^dev_esign[abs(dev_neie[b+j])-1])>0)
				mini=min(mini,dev_h[dev_nein[b+j]]);
		}
		else
			break;
	}
	if(mini!=INT_MAX)
		dev_h[i]=mini+1,*mark=1;
};
void parallelor::prepush(int s,int t,int bw)
{
	cout<<"begin"<<endl;
	int W=WD+1;
	int*dev_h,*h,*dev_v,*v,*dev_neie,*neie,*dev_nein,*nein;
	int*dev_esign,*esign;
	int *dev_emark,*emark,*mark,*dev_mark;
	int *minarray;
	int *dev_st,*dev_te;
	int* st,*te;
	h=new int[W*pnodesize*LY];
	v=new int[W*pnodesize*LY];
	mark=new int;
	vector<vector<int>>rawneie(W*pnodesize,vector<int>());
	vector<vector<int>>rawnein(W*pnodesize,vector<int>());
	for(int i=0;i<edges.size();i++)
		for(int j=0;j<W-1;j++)
		{
			int s=edges[i].s*W+j;
			int t=edges[i].t*W+j+1;
			rawneie[s].push_back(i+1);
			rawneie[t].push_back(-(i+1));
			rawnein[s].push_back(t);
			rawnein[t].push_back(s);
		}
	int max=0;
	for(int i=0;i<rawnein.size();i++)
		if(rawnein[i].size()>max)max=rawnein[i].size();
	max++;
	neie=new int[W*pnodesize*max*LY];
	nein=new int[W*pnodesize*max*LY];
	for(int k=0;k<LY;k++)
		{
		int startn=k*W*pnodesize;
		int starte=k*edges.size();
		for(int i=0;i<W*pnodesize;i++)
			{
				for(int j=0;j<max;j++)
				{
					if(j<rawneie[i].size())
						{
							if(rawneie[i][j]<0)
								neie[(startn+i)*max+j]=rawneie[i][j]-starte;
							else
								neie[(startn+i)*max+j]=rawneie[i][j]+starte;
							nein[(startn+i)*max+j]=rawnein[i][j]+startn;
						}
					else
						{
							neie[(startn+i)*max+j]=INT_MAX;
							nein[(startn+i)*max+j]=INT_MAX;
						}
				}
			}
		}
	emark=new int[LY*edges.size()];
	esign=new int[LY*edges.size()];
	st=new int[LY*edges.size()];
	te=new int[LY*edges.size()];
	for(int i=0;i<LY*edges.size();i++)
		emark[i]=INT_MAX;//0
	for(int k=0;k<LY;k++)
		for(int i=0;i<edges.size();i++)
			{
				st[i+k*edges.size()]=edges[i].s*W+k*W*pnodesize;
				te[i+k*edges.size()]=edges[i].t*W+k*W*pnodesize;
				esign[i+k*edges.size()]=edges[i].s*W+k*W*pnodesize+1;
			}
	for(int i=0;i<W*LY*pnodesize;i++)
		{
			h[i]=0;
			v[i]=0;
		}
	for(int k=0;k<LY;k++)
		{
		for(int i=0;i<edges.size();i++)
			if(edges[i].s==s)
				{
				v[k*W*pnodesize+W*edges[i].t+1]=1;
				esign[k*edges.size()+i]*=-1;
				}
		}
	for(int k=0;k<LY;k++)
		{
		int start=k*W*pnodesize;
		for(int i=s*W;i<s*W+W;i++)
			h[i+start]=WD;
		}
	hipMalloc((void**)&dev_h,LY*W*pnodesize*sizeof(int));
	hipMalloc((void**)&dev_mark,sizeof(int));
	hipMalloc((void**)&dev_v,LY*W*pnodesize*sizeof(int));
	hipMalloc((void**)&dev_neie,LY*W*max*pnodesize*sizeof(int));
	hipMalloc((void**)&dev_nein,LY*W*max*pnodesize*sizeof(int));
	hipMalloc((void**)&dev_esign,LY*edges.size()*sizeof(int));
	hipMalloc((void**)&dev_emark,LY*edges.size()*sizeof(int));
	hipMalloc((void**)&dev_st,LY*edges.size()*sizeof(int));
	hipMalloc((void**)&dev_te,LY*edges.size()*sizeof(int));
	hipMemcpy(dev_h,h,LY*W*pnodesize*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_v,v,LY*W*pnodesize*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_mark,mark,sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_neie,neie,LY*W*max*pnodesize*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_nein,nein,LY*W*max*pnodesize*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_esign,esign,LY*edges.size()*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_emark,emark,LY*edges.size()*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_st,st,LY*edges.size()*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_te,te,LY*edges.size()*sizeof(int),hipMemcpyHostToDevice);
	*mark=1;
	int time=0;
	cout<<"max is "<<max<<endl;
	time_t start,end;
	start=clock();
	while(*mark>0)
	//for(int i=0;i<3;i++)
	{
		//cout<<"************"<<endl;
		*mark=0;
		hipMemcpy(dev_mark,mark,sizeof(int),hipMemcpyHostToDevice);
		push<<<LY*W*pnodesize/WORK_SIZE+1,WORK_SIZE>>>(dev_h,dev_v,dev_esign,dev_emark,dev_neie,dev_nein,W*pnodesize,max,W,s,t,dev_mark);
		aggregate2<<<LY*edges.size()/WORK_SIZE+1,WORK_SIZE>>>(dev_esign,dev_v,dev_emark,W,edges.size());
		//aggregate3<<<LY*edges.size()/WORK_SIZE+1,WORK_SIZE>>>(dev_esign,dev_v,dev_emark,dev_st,dev_te,dev_h,W,edges.size());
		/*hipMemcpy(emark,dev_emark,LY*edges.size()*sizeof(int),hipMemcpyDeviceToHost);
		for(int i=0;i<LY*edges.size();i++)
			if(emark[i]>0)
				cout<<"gota... "<<i<<"s:"<<st[i]<<" "<<te[i]<<" "<<emark[i]<<endl;*/
		//relable<<<LY*W*pnodesize/WORK_SIZE+1,WORK_SIZE>>>(dev_h,dev_v,W*pnodesize,dev_mark,dev_nein,dev_neie,dev_esign,max,W,s,t);
		//aggregate2<<<LY*edges.size()/WORK_SIZE+1,WORK_SIZE>>>(dev_esign,dev_v,dev_emark,W,edges.size(),W*pnodesize,dev_mark);
		hipMemcpy(mark,dev_mark,sizeof(int),hipMemcpyDeviceToHost);
		/*hipMemcpy(v,dev_v,LY*W*pnodesize*sizeof(int),hipMemcpyDeviceToHost);
		hipMemcpy(h,dev_h,LY*W*pnodesize*sizeof(int),hipMemcpyDeviceToHost);
		hipMemcpy(esign,dev_esign,LY*edges.size()*sizeof(int),hipMemcpyDeviceToHost);
		int flow=0;
		for(int i=0;i<LY*W*pnodesize;i++)
			if(v[i]!=0)
				{
					int bi=i%(W*pnodesize);
					if(bi/W==t)flow+=v[i];
					cout<<i/(W*pnodesize)<<" "<<bi<<" "<<bi/W<<" "<<bi%W<<" "<<h[i]<<" "<<v[i]<<endl;
					/*if(i==319)
					{
						for(int j=0;j<max;j++)
							if(nein[i*max+j]<INT_MAX)
								cout<<neie[i*max+j]<<" "<<esign[abs(neie[i*max+j])-1]<<" "<<h[nein[i*max+j]]<<endl;
					}
				}*/
		time++;
	}
	end=clock();
	cout<<"GPU time is: "<<end-start<<endl;
	hipMemcpy(v,dev_v,LY*W*pnodesize*sizeof(int),hipMemcpyDeviceToHost);
	hipMemcpy(h,dev_h,LY*W*pnodesize*sizeof(int),hipMemcpyDeviceToHost);
	int flow=0;
	for(int i=0;i<LY*W*pnodesize;i++)
		if(v[i]!=0)
			{
				int bi=i%(W*pnodesize);
				if(bi/W==t)flow+=v[i];
				//cout<<i/(W*pnodesize)<<" "<<bi<<" "<<bi/W<<" "<<bi%W<<" "<<h[i]<<" "<<v[i]<<endl;
			}
	hipMemcpy(esign,dev_esign,LY*edges.size()*sizeof(int),hipMemcpyDeviceToHost);
	int count=0;
	for(int i=0;i<edges.size()*LY;i++)
		if(esign[i]<0)
			count++;
	cout<<"resort"<<endl;
	/*for(int i=0;i<edges.size()*LY;i++)
		{
			if(esign[i]<0)
			{
				int sorce=edges[i].t*W;
				if(sorce/W==t)
				{
					int pre=edges[i].s*W;
					cout<<pre<<" ";
					while((pre/W)!=s)
					{
						int flag=0;
						for(int h=0;h<W;h++)
						{
							pre++;
							for(int k=0;k<max;k++)
								{
									if(nein[pre*max+k]<INT_MAX)
										if(esign[abs(neie[pre*max+k])-1]<0&&neie[pre*max+k]<0)
										{
											esign[abs(neie[pre*max+k])-1]*=-1;
											pre=edges[abs(neie[pre*max+k])-1].s*W;
											cout<<pre<<" ";
											flag=1;
										}
										if(flag==1)break;
								}
							if(flag==1)break;
						}
					}
					cout<<endl;
				}
			}
		}*/
	cout<<"flow is"<<flow<<endl;
	cout<<"count is "<<count<<endl;
	cout<<"die is "<<time<<endl;
	delete[] h;
	delete[] minarray;
	delete[] v;
	delete[] mark;
	delete[] neie;
	delete[] nein;
	delete[]emark;
	delete[]esign;
	hipFree(dev_h);
	hipFree(dev_mark);
	hipFree(dev_v);
	hipFree(dev_neie);
	hipFree(dev_nein);
	hipFree(dev_esign);
	hipFree(dev_emark);
};
